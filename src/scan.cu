#include "hip/hip_runtime.h"
#include "scan.cuh"

#include "mesh.cuh"
#include "framebuffer.cuh"

#include <iostream>

WorkingQueue::WorkingQueue(uint2 fSize){
    this->capacity = fSize.x * fSize.y;
    hipMalloc(&this->elements, this->capacity * sizeof(QueueElement));
}

WorkingQueue::~WorkingQueue(){
    hipFree(elements);
}

__global__ void computePixelQueue(int k, CudaMesh mesh, WorkingQueue queue, size_t* size) {
    //printf("T [%d, %d, %d]>", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("T [%u, %u, %u]> %lu %u\n", threadIdx.x, threadIdx.y, threadIdx.z, mesh.size, k);

    int triIdx = blockIdx.x;
    if (triIdx >= mesh.size) return;

    Triangle3D tri = mesh.triangles[triIdx];
    Triangle2Di uv = mesh.uvProj[triIdx];

    uint minX = min(uv.v[0].x, min(uv.v[1].x, uv.v[2].x));
    uint minY = min(uv.v[0].y, min(uv.v[1].y, uv.v[2].y));
    uint maxX = max(uv.v[0].x, max(uv.v[1].x, uv.v[2].x));
    uint maxY = max(uv.v[0].y, max(uv.v[1].y, uv.v[2].y));

    printf("X: %d %d\n", minX, maxX);
    printf("Y: %d %d\n", minY, maxY);

    uint2 pos;

    for (pos.x = threadIdx.x + minX; pos.x <= maxX; pos.x += blockDim.x) {
        for (pos.y = threadIdx.y + minY; pos.y <= maxY; pos.y += blockDim.y) {

            float3 barCoord = getBarCoord(pos, uv);

            if (barCoordInside(barCoord)) {
                int index = atomicAdd((unsigned long long*) &queue.size, (unsigned long long) 1);
                queue.elements[queue.size] = {
                    pos,
                    applyBarCoord(barCoord, tri),
                    mesh.normals[triIdx]
                };

                printf("%d %d added to queue\n", pos.x, pos.y);
            }
        }
    }
}
