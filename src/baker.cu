#include "baker.cuh"

#include "scan.cuh"


void computeGI(CpuMesh& mesh, Framebuffer& f){
    WorkingQueue queue(f.size);
    CudaMesh cudaMesh(mesh);

    size_t* size;
    hipMallocManaged(&size, sizeof(size_t));
    *size = 0;

    printf("Mesh size: %d\n", mesh.size); 

    computePixelQueue<<<2,3>>>(666, cudaMesh, queue, size);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    printf("QueueSize: %d\n", *size);

    std::vector<QueueElement> cpuQueue(queue.size);
    hipMemcpy(queue.elements, cpuQueue.data(), queue.size * sizeof(QueueElement), hipMemcpyDeviceToHost);

    return;
}