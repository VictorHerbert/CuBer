#include "hip/hip_runtime.h"
#include "baker.cuh"

#include "scan.cuh"


void computeLightmap(CpuMesh& mesh, Framebuffer& f){
    WorkingQueue queue(f.size);
    CudaMesh cudaMesh(mesh);

    size_t* sizePtr;
    hipMallocManaged(&sizePtr, sizeof(size_t));
    *sizePtr = 0;

    computePixelQueue<<<2,3>>>(cudaMesh, queue, sizePtr);
    hipDeviceSynchronize();

    queue.size = *sizePtr;

    std::vector<QueueElement> cpuQueue(queue.size);
    hipMemcpy(cpuQueue.data(), queue.elements, queue.size * sizeof(QueueElement), hipMemcpyDeviceToHost);    

    for(QueueElement el : cpuQueue){
        f.putPixel(make_int2(el.uv.x, el.uv.y), {255, 255, 255});
    }


    cudaMesh.free();
    queue.free();

    return;
}